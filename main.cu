#include <fstream>
#include <iostream>

#include <radium/camera.cuh>
#include <radium/image.cuh>

int main() {
  std::cout << "Hello, World!" << std::endl;

  const auto width = 16;
  const auto height = 9;

  const auto aspect_ratio = width / height;

  Radium::Camera camera(aspect_ratio);

  Radium::Image image(1920, 1080, camera);

  image.render();

  hipDeviceSynchronize();

  image.save("image.ppm");

  return 0;
}